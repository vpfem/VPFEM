#include "StiffnessMatrix.h"

StiffnessMatrix::StiffnessMatrix(double* mat, Geometry& geo,   int n)
    : material(mat), geometry(&geo), numberOfIntegrationPoint(n)
{
    numberOfElements = geometry->get_numberOfElementsG();
    nipSquared = numberOfIntegrationPoint*numberOfIntegrationPoint;
    simulationSize = numberOfElements*nipSquared;
    // integration points
    hipMallocManaged(&integrationNode, numberOfIntegrationPoint*sizeof(double));
    hipMallocManaged(&integrationPos, numberOfIntegrationPoint*dimention*numberOfIntegrationPoint*sizeof(  int));
    hipMallocManaged(&integrationWeight, numberOfIntegrationPoint*sizeof(double));
    integrationPoint();
    INFO("StiffnessMatrix Created by CPU");
};

StiffnessMatrix::~StiffnessMatrix()
{
    INFO("StiffnessMatrix Deleted by CPU");
    hipFree(integrationNode);
    hipFree(integrationPos);
    hipFree(integrationWeight);
};

void StiffnessMatrix::integrationPoint()
// Creats the integration points
// XI = integrationNode[integrationPos[i]] YI = integrationNode[integrationPos[i+1]] 
{

      int counter = 0;
    for (  int i = 0; i < numberOfIntegrationPoint; i++)
        for (  int j = 0; j < numberOfIntegrationPoint; j++)
        {
            integrationPos[counter++] = i;
            integrationPos[counter++] = j;
        };
    if (numberOfIntegrationPoint == 1) {
        integrationNode[0] = 0; integrationWeight[0] = 2.0;
    } else if (numberOfIntegrationPoint == 2) {
        integrationNode[0] = -0.57735; integrationWeight[0] = 1.0;
        integrationNode[1] =  0.57735; integrationWeight[1] = 1.0;
    } else if (numberOfIntegrationPoint == 3) {
        integrationNode[0] = -0.774596; integrationWeight[0] = 0.555556;
        integrationNode[1] =  0.0     ; integrationWeight[1] = 0.888889;
        integrationNode[2] =  0.774596; integrationWeight[2] = 0.555556;
    } else if (numberOfIntegrationPoint == 4) {
        integrationNode[0] = -0.861136; integrationWeight[0] = 0.347855;
        integrationNode[1] = -0.339981; integrationWeight[1] = 0.652145;
        integrationNode[2] =  0.339981; integrationWeight[2] = 0.652145;
        integrationNode[3] =  0.861136; integrationWeight[3] = 0.347855;
    } else if (numberOfIntegrationPoint == 5) {
        integrationNode[0] = -0.90618;  integrationWeight[0] = 0.236927;
        integrationNode[1] = -0.538469; integrationWeight[1] = 0.478629;
        integrationNode[2] =  0.0     ; integrationWeight[2] = 0.568889;
        integrationNode[3] =  0.538469; integrationWeight[3] = 0.478629;
        integrationNode[4] =  0.90618;  integrationWeight[4] = 0.236927;
    } else {
        printf("Integration points more than five is under construction");
    }
};


